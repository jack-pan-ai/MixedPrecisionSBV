#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <magma_v2.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <mpi.h>
#include <vector>
#include <iostream>
#include <cmath>

#include "gpu_operations.h"
#include "gpu_covariance.h"
#include "prediction.h"
#include "error_checking.h"

// Custom functor for strided access
struct strided_access
{
    double* ptr;
    int stride;
    strided_access(double* _ptr, int _stride) : ptr(_ptr), stride(_stride) {}
    __host__ __device__
    double operator()(int i) const { return ptr[i * stride]; }
};


// Function to perform prediction on the GPU
void performPredictionOnGPU(const GpuData &gpuData, const std::vector<double> &theta, const Opts &opts)
{
    int rank;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    // Set the GPU
    checkCudaError(hipSetDevice(opts.gpu_id));

    // set the stream
    hipStream_t stream=opts.stream;
    magma_queue_t queue = opts.queue;
    
    size_t batchCount = gpuData.ldda_locs.size() - 1;
    magma_int_t *dinfo_magma = gpuData.dinfo_magma;
    int *d_ldda_locs = gpuData.d_ldda_locs;
    int *d_ldda_neighbors = gpuData.d_ldda_neighbors;
    int *d_ldda_cov = gpuData.d_ldda_cov;
    int *d_ldda_cross_cov = gpuData.d_ldda_cross_cov;
    int *d_ldda_conditioning_cov = gpuData.d_ldda_conditioning_cov;
    int *d_lda_locs = gpuData.d_lda_locs;
    int *d_lda_locs_neighbors = gpuData.d_lda_locs_neighbors;
    int *d_const1 = gpuData.d_const1;
    magma_int_t max_m = gpuData.max_m;
    magma_int_t max_n1 = gpuData.max_n1;
    magma_int_t max_n2 = gpuData.max_n2;

    // copy the data from the device to the device
    checkCudaError(hipMemcpy(gpuData.d_observations_neighbors_copy_device, 
                                   gpuData.d_observations_neighbors_device, 
                                   gpuData.total_observations_neighbors_size, 
                                   hipMemcpyDeviceToDevice));
    checkCudaError(hipMemcpy(gpuData.d_observations_copy_device, 
                                   gpuData.d_observations_device, 
                                   gpuData.total_observations_points_size, 
                                   hipMemcpyDeviceToDevice));

    // Use the data on the GPU for computation
    // 1. generate the covariance matrix, cross covariance matrix, conditioning covariance matrix
    // take record of the time
    for (size_t i = 0; i < batchCount; ++i)
    {   
        RBF_matcov(gpuData.h_locs_array[i],
                    gpuData.lda_locs[i], 1, gpuData.total_locs_num_device,
                    gpuData.h_locs_array[i],
                    gpuData.lda_locs[i], 1, gpuData.total_locs_num_device,
                    gpuData.h_cov_array[i], gpuData.ldda_cov[i], gpuData.lda_locs[i],
                    opts.dim, theta, true, stream);
        RBF_matcov(gpuData.h_locs_neighbors_array[i], 
                    gpuData.lda_locs_neighbors[i], 1, gpuData.total_locs_neighbors_num_device,
                    gpuData.h_locs_array[i],
                    gpuData.lda_locs[i], 1, gpuData.total_locs_num_device,
                    gpuData.h_cross_cov_array[i], gpuData.ldda_cross_cov[i], gpuData.lda_locs[i],
                    opts.dim, theta, false, stream);
        RBF_matcov(gpuData.h_locs_neighbors_array[i],
                    gpuData.lda_locs_neighbors[i], 1, gpuData.total_locs_neighbors_num_device,
                    gpuData.h_locs_neighbors_array[i], 
                    gpuData.lda_locs_neighbors[i], 1, gpuData.total_locs_neighbors_num_device,
                    gpuData.h_conditioning_cov_array[i], gpuData.ldda_conditioning_cov[i], gpuData.lda_locs_neighbors[i],
                    opts.dim, theta, true, stream);
        // Synchronize to make sure the kernel has finished
        checkCudaError(hipStreamSynchronize(stream));
    }    
    
    // 2. perform the computation
    // 2.1 compute the correction term for mean and variance (i.e., Schur complement)
    magma_dpotrf_vbatched(MagmaLower, d_lda_locs_neighbors,
                        gpuData.d_conditioning_cov_array, d_ldda_conditioning_cov,
                        dinfo_magma, batchCount, queue);
    // trsm
    magmablas_dtrsm_vbatched_max_nocheck(MagmaLeft, MagmaLower, MagmaNoTrans, MagmaNonUnit, 
                        max_m, max_n1, 
                        d_lda_locs_neighbors, d_lda_locs,
                        1.,
                        gpuData.d_conditioning_cov_array, d_ldda_conditioning_cov,
                        gpuData.d_cross_cov_array, d_ldda_cross_cov,
                        batchCount, queue);
    magmablas_dtrsm_vbatched_max_nocheck(MagmaLeft, MagmaLower, MagmaNoTrans, MagmaNonUnit, 
                        max_m, max_n2, 
                        d_lda_locs_neighbors, d_const1,
                        1.,
                        gpuData.d_conditioning_cov_array, d_ldda_conditioning_cov,
                        gpuData.d_observations_neighbors_copy_array, d_ldda_neighbors,
                        batchCount, queue);
    // gemm
    magmablas_dgemm_vbatched_max_nocheck(MagmaTrans, MagmaNoTrans,
                             d_lda_locs, d_lda_locs, d_lda_locs_neighbors,
                             1, gpuData.d_cross_cov_array, d_ldda_cross_cov,
                                gpuData.d_cross_cov_array, d_ldda_cross_cov,
                             0, gpuData.d_cov_correction_array, d_ldda_cov,
                             batchCount, 
                             max_n1, max_n1, max_m, 
                             queue);
    magmablas_dgemm_vbatched_max_nocheck(MagmaTrans, MagmaNoTrans,
                             d_lda_locs, d_const1, d_lda_locs_neighbors,
                             1, gpuData.d_cross_cov_array, d_ldda_cross_cov,
                                gpuData.d_observations_neighbors_copy_array, d_ldda_neighbors,
                             0, gpuData.d_mu_correction_array, d_ldda_locs,
                             batchCount, 
                             max_n1, max_n2, max_m,
                             queue);
    checkCudaError(hipStreamSynchronize(stream));
    // 2.2 compute the conditional mean and variance
    for (size_t i = 0; i < batchCount; ++i){
        // compute conditional variance
        magmablas_dgeadd(gpuData.lda_locs[i], gpuData.lda_locs[i],
                        -1.,
                        gpuData.h_cov_correction_array[i], gpuData.ldda_locs[i], 
                        gpuData.h_cov_array[i], gpuData.ldda_cov[i],
                        queue);
        // compute conditional mean
        magmablas_dgeadd(gpuData.lda_locs[i], 1,
                        -1.,
                        gpuData.h_mu_correction_array[i], gpuData.ldda_locs[i], 
                        gpuData.h_observations_copy_array[i], gpuData.ldda_locs[i],
                        queue);
    }
    checkCudaError(hipStreamSynchronize(stream));

    // 2.3 generate by reparametrization
    // magma_dprint_gpu(gpuData.lda_locs[0], gpuData.lda_locs[0], gpuData.h_cov_array[0], gpuData.ldda_cov[0], queue);
    checkMagmaError(magma_dpotrf_vbatched(
            MagmaLower, d_lda_locs,
            gpuData.d_cov_array, d_ldda_cov,
            dinfo_magma, batchCount, queue));
    checkCudaError(hipStreamSynchronize(stream));

    // do the conditional simulation
    // // allocate the memory for the conditional simulation
    int num_observations = gpuData.total_observations_points_size/sizeof(double);
    double *d_observations_conditional_device;
    double **h_observations_conditional_array = new double*[opts.numBlocksPerProcess_test * opts.num_simulations];
    double **_d_observations_conditional_array_temp;
    checkCudaError(hipMalloc(&d_observations_conditional_device, num_observations * sizeof(double) * opts.num_simulations));
    checkCudaError(hipMemset(d_observations_conditional_device, 0, num_observations * sizeof(double) * opts.num_simulations));
    checkCudaError(hipMalloc(&_d_observations_conditional_array_temp, opts.numBlocksPerProcess_test * sizeof(double *)));
    int offset_outer_data = 0;
    int offset_outer_array   = 0;
    for (size_t i = 0; i < opts.num_simulations; ++i){
        int offset_inner = 0;
        for (size_t j = 0; j < opts.numBlocksPerProcess_test; ++j){
            h_observations_conditional_array[j + offset_outer_array] = d_observations_conditional_device + offset_outer_data + offset_inner;
            offset_inner += gpuData.ldda_locs[j];
        }
        offset_outer_data += num_observations;
        offset_outer_array += opts.numBlocksPerProcess_test;
    }
    // generate the random noise - h_mu_correction_array
    for (size_t i = 0; i < opts.num_simulations; ++i){
        // generate the random noise,
        // copy h_observations_conditional_array to the device
        checkCudaError(hipMemcpy(_d_observations_conditional_array_temp, h_observations_conditional_array + i * opts.numBlocksPerProcess_test, opts.numBlocksPerProcess_test * sizeof(double *), hipMemcpyHostToDevice));
        // // gpuData.h_mu_correction_array[0] means we generate noise for all blocks
        generate_normal(gpuData.h_mu_correction_array[0], num_observations, 0, 1, rank * opts.num_simulations + i, stream);
        checkCudaError(hipStreamSynchronize(stream));
        // cholesky factor gemm
        magmablas_dgemm_vbatched_max_nocheck(MagmaNoTrans, MagmaNoTrans,
                             d_lda_locs, d_const1, d_lda_locs,
                             1, gpuData.d_cov_array, d_ldda_cov,
                                gpuData.d_mu_correction_array, d_ldda_locs,
                             0, _d_observations_conditional_array_temp, d_ldda_locs,
                             batchCount, 
                             max_n1, max_n2, max_n1,
                             queue);
        checkCudaError(hipStreamSynchronize(stream));
        // error term
        magma_daxpy(num_observations, -1.,
                    gpuData.d_observations_copy_device, 1,
                    d_observations_conditional_device + num_observations * i, 1,
                    queue);
        checkCudaError(hipStreamSynchronize(stream));
    }

    // Calculate mean and standard deviation using Thrust
    thrust::device_vector<double> d_mean(num_observations);
    thrust::device_vector<double> d_stddev(num_observations);

    // Calculate mean
    for (int i = 0; i < num_observations; ++i) {
        strided_access sa(d_observations_conditional_device + i, num_observations);
        double sum = thrust::reduce(
            thrust::cuda::par.on(stream),
            thrust::make_transform_iterator(thrust::counting_iterator<int>(0), sa),
            thrust::make_transform_iterator(thrust::counting_iterator<int>(opts.num_simulations), sa),
            0.0, 
            thrust::plus<double>()
        );
        d_mean[i] = sum / opts.num_simulations;
    }

    // Calculate standard deviation
    for (int i = 0; i < num_observations; ++i) {
        strided_access sa(d_observations_conditional_device + i, num_observations);
        double mean = d_mean[i];
        double sum_sq = thrust::transform_reduce(
            thrust::cuda::par.on(stream),
            thrust::make_transform_iterator(thrust::counting_iterator<int>(0), sa),
            thrust::make_transform_iterator(thrust::counting_iterator<int>(opts.num_simulations), sa),
            [mean] __device__ (double x) { return (x - mean) * (x - mean); },
            0.0,
            thrust::plus<double>()
        );
        d_stddev[i] = sqrt(sum_sq / (opts.num_simulations - 1));
    }

    // to calculate the SSPE (sum of squared prediction errors)
    double sspe_total = 0;
    double sspe_local = thrust::transform_reduce(
        thrust::cuda::par.on(stream),
        d_mean.begin(),
        d_mean.end(),
        [] __device__ (double x) { return x * x; },
        0.0,
        thrust::plus<double>()
    );
    MPI_Allreduce(&sspe_local, &sspe_total, 1, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);
    double mspe = sspe_total / opts.numPointsTotal_test;
    if (rank == 0){
        std::cout << "MSPE: " << mspe << std::endl;
        std::cout << "--------------------------------" << std::endl;
    }


    //print the mean and standard deviation
    if (rank == 0 && opts.print){
        int offset = 0;
        int count_within_interval = 0;
        int total_points = 0;
        std::cout << "numBlocksPerProcess_test: " << opts.numBlocksPerProcess_test << std::endl;
        for (int i = 0; i < opts.numBlocksPerProcess_test; ++i){
            for (int j = 0; j < gpuData.lda_locs[i]; ++j){
                double mean = d_mean[offset + j];
                double stddev = d_stddev[offset + j];
                double lower_bound = mean - 1.96 * stddev;
                double upper_bound = mean + 1.96 * stddev;
                if (lower_bound <= 0 && upper_bound >= 0) {
                    count_within_interval++;
                }
                total_points++;
                std::cout << "mean[" << i << "][" << j << "]: " << mean << ", stddev[" << i << "][" << j << "]: " << stddev << std::endl;
            }
            offset += gpuData.ldda_locs[i];
        }
        double percentage_within_interval = (static_cast<double>(count_within_interval) / total_points) * 100;
        std::cout << "Percentage of .95 intervals containing 0: " << percentage_within_interval << "%" << std::endl;
    }

    // free d_observations_conditional_device
    hipFree(d_observations_conditional_device);
    hipFree(_d_observations_conditional_array_temp);
}


